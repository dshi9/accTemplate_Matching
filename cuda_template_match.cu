#include "hip/hip_runtime.h"
//Author: Dongwei Shi
//Created: 06/15/2016
//Description: this program is for template matching with cuda. The program is expected to template match several template simutaneously

#include <stdio.h>
#include <iostream>
#include <math.h>
#include <vector>
#include <unistd.h>
#include <string>
#include <opencv2/opencv.hpp>
#include <opencv2/nonfree/features2d.hpp>
#include </usr/local/cuda-8.0/include/hip/hip_runtime.h>
#include </usr/local/cuda-8.0/include/hipfft/hipfft.h>
#include </usr/local/cuda-8.0/include/hipfft/hipfft.h>



#define KERNEL_WIDTH 31
#define KERNEL_RADIUS (KERNEL_WIDTH/2)
#define TILE_WIDTH (33-KERNEL_WIDTH)
#define BLK_SIZE (TILE_WIDTH+KERNEL_WIDTH-1)
#define TMP_NUM 8


#define ACCURATE_MODE KERNEL_WIDTH
#define SPEED_MODE 1
#define RECORD 0
#define CROP_PARAM 2.2
using namespace std;
using namespace cv;

//global image and templates
Mat img, gray_img, prev_img;
Mat templs[TMP_NUM];
Mat img_vec[TMP_NUM];
Point kpt_vec[TMP_NUM];
Point ext_vec[TMP_NUM];
vector<Point2f > corners;
int dis[TMP_NUM];

//deviceKernel for storing the templates 
__constant__ float deviceKernel[TMP_NUM*KERNEL_WIDTH*KERNEL_WIDTH];
///////////////////////////////////////////////////////////////////
/* conv2d
 *      Description: This funtion is CUDA kernel. Where perform the 2D convolution of the images and templates.
 *                   Using CV_TM_CCOEFF_NORMED method for template matching. Simutaneously perform 2D convolution
 *                   on several images with specific templates.
 *      Input: A -- the input data of images
 *             x_size -- the image width
 *             y_size -- the image height
 *             template_num -- the total templates need to be matched.
 *      Output: B -- the convolution results of the images.
 *      
 * 
*/
///////////////////////////////////////////////////////////////////
__global__ void conv2d(float* A, float* B, const int x_size, const int y_size, const int template_num)
{
   //allocated shared memory for storing the image
    __shared__ float Nds[BLK_SIZE][BLK_SIZE];
    int tx = threadIdx.x;
    int ty = threadIdx.y;


    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;

    int x_out = bx*TILE_WIDTH + tx;
    int y_out = by*TILE_WIDTH + ty;
    
    int x_in = x_out - KERNEL_RADIUS;
    int y_in = y_out - KERNEL_RADIUS;
    float res = 0.0;
    float templ_res = 0.0;
    float img_res = 0.0;
    //copy the image to the shared memeory 
    if((x_in>=0) && (x_in<x_size) && (y_in>=0) && (y_in<y_size) && (bz>=0) && (bz<template_num) )
    {
        Nds[ty][tx] = A[bz*x_size*y_size + y_in*x_size + x_in];
    }
    else
    {
        Nds[ty][tx] = 0.0;
    }
    __syncthreads();

    //perform convolution below using CV_TM_CCOEFF_NORMED method for template matching
    if( (tx<TILE_WIDTH) && (ty<TILE_WIDTH) && (x_out<x_size) && (y_out<y_size) && (bz>=0) && (bz<template_num))
    {
            res = 0.0;
            templ_res = 0.0;
            img_res = 0.0;
            for( int idx_y=0; idx_y<KERNEL_WIDTH; idx_y++ )
            {
                for( int idx_x=0; idx_x<SPEED_MODE; idx_x++ )
                {
                    
                    templ_res += pow(deviceKernel[bz*KERNEL_WIDTH*KERNEL_WIDTH+idx_y*KERNEL_WIDTH+idx_x],2);
                    img_res += pow(Nds[ty+idx_y][tx+idx_x],2);
                    res += Nds[ty+idx_y][tx+idx_x] * deviceKernel[bz*KERNEL_WIDTH*KERNEL_WIDTH+idx_y*KERNEL_WIDTH+idx_x];
                    
    
                }
            }
            //copy the result into the output data
            __syncthreads();
            if((x_out<x_size) && (y_out<y_size) && (bz<template_num))
            {
                B[bz*x_size*y_size + y_out*x_size + x_out] = res/sqrt(templ_res*img_res);
            }
            __syncthreads();
        
    }
   
}
///////////////////////////////////////////////////////////////////
/* cuda_tp_img
 *      Description: This function use for preparation step for the 
 *                   cuda kernel. It is allocate several memory space
 *                   on both GPU and CPU. It also be used to select the
 *                   peak value of the convolution results  
 *      Input: templates number -- the total number of templates that need to
 *                                 be matched.
 *      Output: 0 -- success, -1 -- failure
 *      
 * 
*/
///////////////////////////////////////////////////////////////////

int cuda_tp_img(int template_num)
{
   
    //get size of templates and images.
    int x_size = img_vec[0].cols;
    int y_size = img_vec[0].rows;
    int tmp_x_size = KERNEL_WIDTH;//templs[0].cols;
    int tmp_y_size = KERNEL_WIDTH;//templs[0].rows;
    int img_size = x_size * y_size;
    int tmpl_size = tmp_x_size * tmp_y_size;
    
    //allocate a space to store the image intensity
    float* host_img = (float*) malloc(sizeof(float)*img_size*template_num);
    float* host_templ = (float*) malloc(sizeof(float)*tmpl_size*template_num);
    float* gpu_out = (float*) malloc(sizeof(float)*img_size*template_num);

    float* device_img_input;
    float* device_img_output;
  
    //copy the intensity value from image
    for(int img_idx=0; img_idx<template_num; img_idx++)
    {
        for(int y=0; y<y_size; y++)
        {
            for(int x=0; x<x_size; x++)
            {
                Scalar intensity = img_vec[img_idx].at<uchar>(y,x);
                host_img[y*x_size+x + img_idx*img_size] = intensity.val[0];
            }   
         } 
         
    }
    //copy the intensity value from templates
    for(int tmpl_idx=0; tmpl_idx<template_num; tmpl_idx++)
    {
        for(int y=0; y<tmp_y_size; y++)
        {
            for(int x=0; x<tmp_x_size; x++)
            {
                Scalar intensity = templs[tmpl_idx].at<uchar>(y,x);
                host_templ[y*tmp_x_size+x+tmpl_idx*tmpl_size] = intensity.val[0];
            }        
        }
    }
    //allocate memory in cuda global memory
    hipMalloc( (void**)&device_img_input, img_size*sizeof(float)*template_num  );
    hipMalloc( (void**)&device_img_output, img_size*sizeof(float)*template_num );

    hipMemcpy( device_img_input, host_img, img_size*sizeof(float)*template_num, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL( deviceKernel), host_templ, tmpl_size*sizeof(float)*template_num);

    //assign blocks and threads
    dim3 Dimblock(BLK_SIZE, BLK_SIZE, 1);
    dim3 DimGrid(((TILE_WIDTH+x_size)-1/TILE_WIDTH), ((TILE_WIDTH+y_size)-1/TILE_WIDTH),template_num);
    //calling the convolution gpu function
    conv2d <<< DimGrid, Dimblock >>>( device_img_input, device_img_output, x_size, y_size, template_num);
    hipDeviceSynchronize();
    
    hipMemcpy( gpu_out, device_img_output, img_size*sizeof(float)*template_num, hipMemcpyDeviceToHost);
    //Selecting peak value of each image's convolution result and label out on the image.
    float res = 0;
    int y_pos;
    for(int idx=0; idx<template_num; idx++)
    {
        y_pos = 0;
        res = 0;
        for(int y=0; y<y_size; y++)
        {
            for(int x=0; x<x_size; x++)
            {
                
                if(gpu_out[idx*img_size+y*x_size+x]>res)
                {
                    res = gpu_out[idx*img_size+y*x_size+x];
                    y_pos = y;
                }
            }  
        }
        ext_vec[idx].x = kpt_vec[idx].x;
        ext_vec[idx].y = (img.rows/CROP_PARAM)+dis[idx]+y_pos;
        rectangle(img, Point(kpt_vec[idx].x-KERNEL_RADIUS,(img.rows/CROP_PARAM)+dis[idx]+y_pos-KERNEL_RADIUS), Point(kpt_vec[idx].x+KERNEL_RADIUS,(img.rows/CROP_PARAM)+dis[idx]+y_pos+KERNEL_RADIUS), Scalar(0,255,0 ), 1, 4);
        line(img,kpt_vec[idx],Point(kpt_vec[idx].x,(img.rows/CROP_PARAM)+dis[idx]+y_pos),Scalar(0,0,255),1,8,0);
    }

    //Free the allocated memory before    
    hipFree(device_img_input);
    hipFree(device_img_output);
    free(host_img);
    free(host_templ);
    free(gpu_out);
    return 0;
}
/////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char*argv[])
{
    
    //declear varible here
    int template_num;
    int start = 0;
    vector<Point2f > pred_vec;
    vector<Point2f > ref_pred_vec;
    Mat status;
    Mat ref_status;
    Mat err;
    Mat ref_err;
    //VideoWriter video("reflection_matching.avi", CV_FOURCC('M','J','P','G'), 10, Size(800, 600),true);
    
    while(1)
    {
        char filename[256];
        fscanf(stdin, "%s", filename);
        //cout << filename << endl;
        template_num = TMP_NUM;
        img = imread(filename, 1);
        img = img(Rect(30,15,img.cols-65,img.rows-45));
        //imshow("input",img);
        //waitKey(0);
        if(!img.data)
        {
            cout << "Problem loading image !!!" << endl;
            return -1;
        }
        //convert the image to gray scale in order to only have one pointer
        cvtColor(img, gray_img, CV_BGR2GRAY);
        //cropping the image
        
        Mat hf_img = gray_img(Rect(0,0,gray_img.cols,gray_img.rows/CROP_PARAM));

        Mat mask;
        bool useHarrisDetector = false;
        
        goodFeaturesToTrack(hf_img, corners, TMP_NUM, 0.01, 20.0, mask, 3, useHarrisDetector, 0.04);
        //imshow("hf_img", hf_img);
        //waitKey(0);

        if(corners.size() == 0)
        {
            cout << "bad frame" << endl;
            continue;
        }
        Point kpt;
 
        for(int temp_generate_idx = 0; temp_generate_idx<template_num; temp_generate_idx++)
        {   
            kpt = corners[temp_generate_idx];
            //get the predict distance
            dis[temp_generate_idx] = gray_img.rows/CROP_PARAM-kpt.y;

            //boundary check for the images
            if( kpt.x < KERNEL_RADIUS)  
                kpt.x = KERNEL_RADIUS;
            if( kpt.x > (img.cols-KERNEL_WIDTH) )
                kpt.x = img.cols-KERNEL_WIDTH;
            if( kpt.y < KERNEL_RADIUS)
                kpt.y = KERNEL_RADIUS;
            if( kpt.y > ((img.rows/CROP_PARAM+dis[temp_generate_idx])-KERNEL_WIDTH) )
                kpt.y = (img.rows/CROP_PARAM+dis[temp_generate_idx])-KERNEL_WIDTH;

            //label the original feature point of the image
            rectangle(img, Point(kpt.x-KERNEL_RADIUS,kpt.y-KERNEL_RADIUS), Point(kpt.x+KERNEL_RADIUS,kpt.y+KERNEL_RADIUS), Scalar(255,0,0 ), 1, 4);
            Mat curr_tmpl = hf_img(Rect(kpt.x-KERNEL_RADIUS,kpt.y-KERNEL_RADIUS,KERNEL_WIDTH,KERNEL_WIDTH));
            //flip the template in order to find the reflections
            flip(curr_tmpl,templs[temp_generate_idx],0);

            /*
            imshow("img", img);
            waitKey(0);
            printf("%d:%d\n", temp_generate_idx,dis[temp_generate_idx]);
            */

            //cropping the image
            img_vec[temp_generate_idx] = gray_img(Rect(kpt.x-KERNEL_RADIUS,gray_img.rows/CROP_PARAM+dis[temp_generate_idx],KERNEL_WIDTH,gray_img.rows-(gray_img.rows/CROP_PARAM+dis[temp_generate_idx])));
            
            /*
            imshow("temp_img",img_vec[temp_generate_idx]);
            waitKey(0);
            */
            kpt_vec[temp_generate_idx] = kpt;
            
        }
          
        cuda_tp_img(template_num);
        if( start == 0 )
        {
            start = 1;
            prev_img = img;
            continue;
        }
        /////**optical flow track starts here**/////
        calcOpticalFlowPyrLK(prev_img, img, corners, pred_vec, status, err);

        //calcOpticalFlowPyrLK(prev_img, img, ref_corners, ref_pred_vec, ref_status, ref_err);
        prev_img = img;
        //video.write(img);
        //line(img, Point(0,img.rows/CROP_PARAM), Point(img.cols,img.rows/CROP_PARAM), Scalar(110,220,0));
        imshow("img", img);
        waitKey(1);
    }

}
